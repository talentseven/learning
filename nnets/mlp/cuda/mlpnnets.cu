/*

  mlpnnets.cu
  Implementation of feedforward MLP neural networks in CUDA.

  Andrei de A. Formiga, 2012-05-09

 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>

// --- activation functions -----------------------------------------------

// sigmoid activation function
__device__ float asigmoid(float t)
{
    return 1.0f / (1.0f + expf(-t));
}

__device__ float dsigmoid(float output)
{
    return output * (1.0f - output);
}


// --- initialization -----------------------------------------------------

// make randomly generated weights in (0.0, 1.0] be in the
// interval from -max_abs to +max_abs
__global__ void normalize_weights(float *w, float max_abs)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    w[tid] = ((w[tid] - 0.5f) / 0.5f) * max_abs;
}

// random initialization for weights
// w must be an array of floats on the device
void random_weights(float *w, float max_abs, int nweights, long seed)
{
    hiprandGenerator_t gen;

    // create and initialize generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_SEEDED);

    hiprandGenerateUniform(gen, w, nweights);
    normalize_weights<<<1, nweights>>>(w, max_abs);
    hiprandDestroyGenerator(gen);
}

// initialize weights randomly using the supplied generator
// w must be an array of floats on the device
void random_weights_gen(float *w, float max_abs, int nweights, hiprandGenerator_t gen)
{
    hiprandGenerateUniform(gen, w, nweights);
    normalize_weights<<<1, nweights>>>(w, max_abs);
}


// --- forward propagation ------------------------------------------------

