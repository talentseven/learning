#include "hip/hip_runtime.h"
/*

  mlpnnets.cu
  Implementation of feedforward MLP neural networks in CUDA.

  Andrei de A. Formiga, 2012-05-09

 */

#include <stdio.h>
#include <stdlib.h>

#include "mlpnnets.h"


// --- utility functions --------------------------------------------------
inline float* allocateFloatsDev(int n)
{
    float *res;

    if (hipMalloc((void**) &res, n * sizeof(float)) != hipSuccess) {
        return NULL;
    }

    return res;
}

// --- activation functions -----------------------------------------------

// sigmoid activation function
__device__ float asigmoid(float t)
{
    return 1.0f / (1.0f + expf(-t));
}

__device__ float dsigmoid(float output)
{
    return output * (1.0f - output);
}


// --- initialization -----------------------------------------------------

// make randomly generated weights in (0.0, 1.0] be in the
// interval from -max_abs to +max_abs
__global__ void normalize_weights(float *w, float max_abs)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    w[tid] = ((w[tid] - 0.5f) / 0.5f) * max_abs;
}

// random initialization for weights
// w must be an array of floats on the device
void RandomWeights(MLPNetwork *net, float max_abs, long seed)
{
    hiprandGenerator_t gen;

    // create and initialize generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_SEEDED);

    hiprandGenerateUniform(gen, net->d_weights, net->nWeights);
    normalize_weights<<<1, net->nWeights>>>(net->d_weights, max_abs);
    hiprandDestroyGenerator(gen);
}

// initialize weights randomly using the supplied generator
// w must be an array of floats on the device
void RandomWeightsGen(MLPNetwork *net, float max_abs, hiprandGenerator_t gen)
{
    hiprandGenerateUniform(gen, net->d_weights, net->nWeights);
    normalize_weights<<<1, net->nWeights>>>(net->d_weights, max_abs);
}


// --- network construction and management --------------------------------
void DestroyLayer(MLPLayer *layer)
{
    if (layer->d_outs != NULL)
        hipFree(layer->d_outs);

    if (layer->d_deltas != NULL)
        hipFree(layer->d_deltas);

    free(layer);
}

MLPLayer *CreateLayer(int nNeurons, int nNeuronsPrev, int wOffset, int nCases)
{
    MLPLayer *result = (MLPLayer*) calloc(1, sizeof(MLPLayer));

    if (result == NULL)
        return NULL;

    result->nNeurons = nNeurons;

    // allocate outputs and deltas on device
    result->d_outs = allocateFloatsDev(nNeurons * nCases);

    if (result->d_outs == NULL) {
        DestroyLayer(result);
        return NULL;
    }

    // TODO: deltas allocated per case?
    result->d_deltas = allocateFloatsDev(nNeurons * nCases);

    if (result->d_deltas == NULL) {
        DestroyLayer(result);
        return NULL;
    }

    result->weightsPerNeuron = nNeuronsPrev + 1;
    result->weightOffset = wOffset;

    return result;
}

// Create a MLP neural network for execution on the GPU.
// nLayers: number of layers
// neuronsPerLayer: array of ints (size equal to nLayers) with the
//                  number of neurons for each layer
// nCases: Number of input cases to process in parallel
MLPNetwork *CreateNetwork(int nLayers, int *neuronsPerLayer, int nCases)
{
    MLPNetwork *result;

    result = (MLPNetwork*) calloc(1, sizeof(MLPNetwork));
    
    if (result == NULL)
        return NULL;

    result->nCases = nCases;
    
    result->nLayers = nLayers;
    result->layers = (MLPLayer**) calloc(nLayers, sizeof(MLPLayer*));

    if (result->layers == NULL) {
        free(result);
        return NULL;
    }

    // create input layer
    result->layers[0] = CreateLayer(neuronsPerLayer[0], 0, 0, nCases);
    if (result->layers[0] == NULL) {
        DestroyNetwork(result);
        return NULL;
    }

    // create remaining layers, and sum the number of weights
    int nwTotal = 0;
    int nwPrev = neuronsPerLayer[0];        
    for (int i = 1; i < nLayers; ++i) {
        result->layers[i] = CreateLayer(neuronsPerLayer[i], nwPrev, nwTotal, nCases);
        if (result->layers[i] == NULL) {
            DestroyNetwork(result);
            return NULL;
        }

        nwTotal += neuronsPerLayer[i] * (nwPrev + 1);
        nwPrev = neuronsPerLayer[i];        
    }

    result->nWeights = nwTotal;
    result->d_weights = allocateFloatsDev(result->nWeights);

    if (result->d_weights == NULL) {
        DestroyNetwork(result);
        return NULL;
    }        

    return result;
}

void DestroyNetwork(MLPNetwork *net)
{
    if (net->d_weights != NULL)
        hipFree(net->d_weights);

    if (net->layers != NULL) {
        for (int i = 0; i < net->nLayers; ++i)
            if (net->layers[i] != NULL)
                DestroyLayer(net->layers[i]);

        free(net->layers);
    }

    free(net);
}


// --- forward propagation ------------------------------------------------

// calculate outputs of one layer, assuming the previous
// layer was already calculated; the outputs corresponding to
// all input cases are computed in parallel
//
// grid will be <<<Nc, Nn>>> for Nc input cases and Nn neurons in layer
__global__ void forward_layer(float *d_weights, int weightOffset, int weightsPerNeuron,
                              float *d_ins, int neuronsPrev, float *d_outs)
{
    // weightsPerNeuron is always = to neuronsPrev+1
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int ixIn = blockIdx.x * neuronsPrev;
    int toff = weightOffset + (threadIdx.x * weightsPerNeuron);

    // bias input
    float a = d_weights[toff];

    for (int i = 1; i < weightsPerNeuron; ++i)
        a += d_weights[toff + i] * d_ins[ixIn + i-1];

    // TODO: make it possible to use other activation functions?
    // (maybe using templates)
    d_outs[tid] = asigmoid(a);
}

// calculate outputs of one layer using a threshold activation,
// assuming the previous layer was already calculated; the outputs
// corresponding to all input cases are computed in parallel
//
// grid will be <<<Nc, Nn>>> for Nc input cases and Nn neurons in layer
__global__ void forward_layer_threshold(float *d_weights, int weightOffset,
                                        int weightsPerNeuron,
                                        float *d_ins, int neuronsPrev,
                                        float *d_outs)
{
    // weightsPerNeuron is always = to neuronsPrev+1
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int ixIn = blockIdx.x * neuronsPrev;
    int toff = weightOffset + (threadIdx.x * weightsPerNeuron);

    // bias input
    float a = d_weights[toff];

    for (int i = 1; i < weightsPerNeuron; ++i)
        a += d_weights[toff + i] * d_ins[ixIn + i-1];

    // TODO: make it possible to use other activation functions?
    // (maybe using templates)
    d_outs[tid] = (a > 0.0f? 1.0f : 0.0f);
}

// present a vector of input cases to the network nnet and do forward propagation.
// inputs is assumed to be in host memory, and of size equal to N * nnet->nCases,
// where N is the number of inputs to the network
void PresentInputs(MLPNetwork *nnet, float *inputs, int actf)
{
    int nInputs = nnet->layers[0]->nNeurons;

    // copy inputs to layer 0 on network
    hipMemcpy(nnet->layers[0]->d_outs, inputs,
               nInputs * nnet->nCases * sizeof(float),
               hipMemcpyHostToDevice);

    int nn;
    for (int l = 1; l < nnet->nLayers; ++l) {
        nn = nnet->layers[l]->nNeurons;
        if (actf == ACTF_THRESHOLD)
            forward_layer_threshold<<<nnet->nCases, nn>>>(nnet->d_weights,
                                                nnet->layers[l]->weightOffset,
                                                nnet->layers[l]->weightsPerNeuron,
                                                nnet->layers[l-1]->d_outs,
                                                nnet->layers[l-1]->nNeurons,
                                                nnet->layers[l]->d_outs);
        else
            forward_layer<<<nnet->nCases, nn>>>(nnet->d_weights,
                                                nnet->layers[l]->weightOffset,
                                                nnet->layers[l]->weightsPerNeuron,
                                                nnet->layers[l-1]->d_outs,
                                                nnet->layers[l-1]->nNeurons,
                                                nnet->layers[l]->d_outs);
    }
    
}

// Copy the outputs for network nnet, stored in device memory, to
// host memory pointed to by outs. outs must have size equal to N * nnet->nCases,
// where N is the number of output neurons in the network
void CopyNetworkOutputs(MLPNetwork *nnet, float *outs)
{
    MLPLayer *last = nnet->layers[nnet->nLayers-1];
    
    hipMemcpy(outs, last->d_outs,
               last->nNeurons * nnet->nCases * sizeof(float),
               hipMemcpyDeviceToHost);
}

void PrintWeights(MLPNetwork *nnet)
{
    float *h_weights;

    h_weights = (float*) malloc(nnet->nWeights * sizeof(float));

    if (h_weights == NULL) {
        printf("Error allocating host memory to copy weights.\n");
    }
    else {
        // TODO: check hipMemcpy for errors
        hipMemcpy(h_weights, nnet->d_weights, nnet->nWeights * sizeof(float),
                   hipMemcpyDeviceToHost);
        
        for (int i = 0; i < nnet->nWeights; ++i) {
            printf("%4.5f ", h_weights[i]);
        }
        printf("\n");        
    }
}

// return an array of floats with the outputs for layer with index ixLayer
float *GetLayerOutputs(MLPNetwork *nnet, int ixLayer)
{
    int   length = nnet->layers[ixLayer]->nNeurons * nnet->nCases;
    float *result = (float*) malloc(length * sizeof(float));

    if (result == NULL)
        return NULL;

    // TODO: check hipMemcpy for errors
    hipMemcpy(result, nnet->layers[ixLayer]->d_outs,
               length * sizeof(float), hipMemcpyDeviceToHost);

    return result;
}