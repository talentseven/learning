#include "hip/hip_runtime.h"
/*

  mlpnnets.cu
  Implementation of feedforward MLP neural networks in CUDA.

  Andrei de A. Formiga, 2012-05-09

 */

#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>

#include "mlpnnets.h"


// --- utility functions --------------------------------------------------
inline float* allocateFloatsDev(int n)
{
    float *res;

    if (hipMalloc((void**) &res, n * sizeof(float)) != hipSuccess) {
        return NULL;
    }

    return res;
}

// --- activation functions -----------------------------------------------

// sigmoid activation function
__device__ float asigmoid(float t)
{
    return 1.0f / (1.0f + expf(-t));
}

__device__ float dsigmoid(float output)
{
    return output * (1.0f - output);
}


// --- initialization -----------------------------------------------------

// make randomly generated weights in (0.0, 1.0] be in the
// interval from -max_abs to +max_abs
__global__ void normalize_weights(float *w, float max_abs)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    w[tid] = ((w[tid] - 0.5f) / 0.5f) * max_abs;
}

// random initialization for weights
// w must be an array of floats on the device
void RandomWeights(MLPNetwork *net, float max_abs, long seed)
{
    hiprandGenerator_t gen;

    // create and initialize generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_SEEDED);

    hiprandGenerateUniform(gen, net->d_weights, net->nWeights);
    normalize_weights<<<1, net->nWeights>>>(net->d_weights, max_abs);
    hiprandDestroyGenerator(gen);
}

// initialize weights randomly using the supplied generator
// w must be an array of floats on the device
void RandomWeightsGen(MLPNetwork *net, float max_abs, hiprandGenerator_t gen)
{
    hiprandGenerateUniform(gen, net->d_weights, net->nWeights);
    normalize_weights<<<1, net->nWeights>>>(net->d_weights, max_abs);
}


// --- network construction and management --------------------------------
void DestroyLayer(MLPLayer *layer)
{
    if (layer->d_outs != NULL)
        hipFree(layer->d_outs);

    if (layer->d_deltas != NULL)
        hipFree(layer->d_deltas);

    free(layer);
}

MLPLayer *CreateLayer(int nNeurons, int nNeuronsPrev, int wOffset, int nCases)
{
    MLPLayer *result = (MLPLayer*) calloc(1, sizeof(MLPLayer));

    if (result == NULL)
        return NULL;

    result->nNeurons = nNeurons;

    // allocate outputs and deltas on device
    result->d_outs = allocateFloatsDev(nNeurons * nCases);

    if (result->d_outs == NULL) {
        DestroyLayer(result);
        return NULL;
    }

    // TODO: deltas allocated per case?
    result->d_deltas = allocateFloatsDev(nNeurons * nCases);

    if (result->d_deltas == NULL) {
        DestroyLayer(result);
        return NULL;
    }

    result->weightsPerNeuron = nNeuronsPrev + 1;
    result->weightOffset = wOffset;

    return result;
}

// Create a MLP neural network for execution on the GPU.
// nLayers: number of layers
// neuronsPerLayer: array of ints (size equal to nLayers) with the
//                  number of neurons for each layer
// nCases: Number of input cases to process in parallel
MLPNetwork *CreateNetwork(int nLayers, int *neuronsPerLayer, int nCases)
{
    MLPNetwork *result;

    result = (MLPNetwork*) calloc(1, sizeof(MLPNetwork));
    
    if (result == NULL)
        return NULL;

    result->nLayers = nLayers;
    result->layers = (MLPLayer**) calloc(nLayers, sizeof(MLPLayer*));

    if (result->layers == NULL) {
        free(result);
        return NULL;
    }

    // create input layer
    result->layers[0] = CreateLayer(neuronsPerLayer[0], 0, 0, nCases);
    if (result->layers[0] == NULL) {
        DestroyNetwork(result);
        return NULL;
    }

    // create remaining layers, and sum the number of weights
    int nwTotal = 0;
    int nwPrev = neuronsPerLayer[0];        
    for (int i = 1; i < nLayers; ++i) {
        result->layers[i] = CreateLayer(neuronsPerLayer[i], nwPrev, nwTotal, nCases);
        if (result->layers[i] == NULL) {
            DestroyNetwork(result);
            return NULL;
        }

        nwTotal += neuronsPerLayer[i] * (nwPrev + 1);
        nwPrev = neuronsPerLayer[i];        
    }

    result->nWeights = nwTotal;
    result->d_weights = allocateFloatsDev(result->nWeights);

    if (result->d_weights == NULL) {
        DestroyNetwork(result);
        return NULL;
    }        

    return result;
}

void DestroyNetwork(MLPNetwork *net)
{
    if (net->d_weights != NULL)
        hipFree(net->d_weights);

    if (net->layers != NULL) {
        for (int i = 0; i < net->nLayers; ++i)
            if (net->layers[i] != NULL)
                DestroyLayer(net->layers[i]);

        free(net->layers);
    }

    free(net);
}


// --- forward propagation ------------------------------------------------

// calculate outputs of one layer, assuming the previous
// layer was already calculated; the outputs corresponding to
// all input cases are computed in parallel
//
// grid will be <<<Nc, Nn>>> for Nc input cases and Nn neurons in layer
__global__ void forward_layer(float *d_weights, int weightOffset, int weightsPerNeuron,
                              float *d_ins, int neuronsPrev, float *d_outs)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int ixIn = (blockIdx.x * blockDim.x) * neuronsPrev;
    int toff = weightOffset + (threadIdx.x * weightsPerNeuron);

    // bias input
    float a = d_weights[toff];

    for (int i = 1; i < weightsPerNeuron; ++i)
        a += d_weights[toff+i] * d_ins[ixIn];

    // TODO: make it possible to use other activation functions?
    // (maybe using templates)
    d_outs[tid] = asigmoid(a);
}

// present a vector of input cases to the network nnet and do forward propagation.
// inputs is assumed to be in host memory, and of size equal to nnet->nCases
void PresentInputs(MLPNetwork *nnet, float *inputs) // FIX: d_outs in MLPNetwork is for 1 case only!
{
    int nInputs = nnet->layers[0]->nNeurons;

    // copy inputs to layer 0 on network
    hipMemcpy(nnet->layers[0]->d_outs, inputs,
               nInputs * nnet->nCases * sizeof(float),
               hipMemcpyHostToDevice);

    int nn;
    for (int l = 1; l < nnet->nLayers; ++l) {
        nn = nnet->layers[l]->nNeurons;
        forward_layer<<<nnet->nCases, nn>>>(nnet->d_weights,
                                            nnet->layers[l]->weightOffset,
                                            nnet->layers[l]->weightsPerNeuron,
                                            nnet->layers[l-1]->d_outs,
                                            nnet->layers[l-1]->nNeurons,
                                            nnet->layers[l]->d_outs);
    }
    
}

void CopyNetworkOutputs(MLPNetwork *nnet, float *outs, int nCases)
{
}
